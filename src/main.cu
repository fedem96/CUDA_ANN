/*
 ============================================================================
 Name        : CUDA_ANN.cu
 Author      : federico
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA Approximate Nearest Neighbors
 ============================================================================
 */

#include <sstream>
#include <iterator>
#include <iostream>
#include <vector>
#include <chrono>
#include <omp.h>
#include <unistd.h>

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

#include "utils.cpp"
#include "read_sift_dataset.cpp"
#include "search.hpp"
#include "csvfile.h"
#ifdef __HIPCC__
#include "search.cuh"
#endif


/** @brief Evaluate a search algorithm.
  * @param s: search algorithm to evaluate
  * @param queries: samples to search
  * @param groundTruth: for each query, ordered vector of indexes that should return the search method
  * @param numResults: number of queries to make
  * @param mustCheckCorrectness: if true, assures that results of the search are the same of groundTruth
  * @return elapsed time to search for all queries
  *
  * This method evaluates the time needed to a search algorithm to search for all queries in a dataset.
  * The search algorithm must be previously initialized on a dataset.
  */
template <typename T>
std::chrono::duration<double> evaluate(Search<T> *s, T* queries, std::vector <std::vector<int>> &groundTruth, const int &numQueries, int &numResults, bool mustCheckCorrectness = false);
// TODO cambiare signature di questa funzione: il groundTruth non è necessario quando non si vuole anche controllare la correttezza dell'algoritmo

/** @brief Check for correcteness
  * @param groundTruth: for each query, ordered vector of indexes that should return the search method
  * @param nnAllIndexes: for each query, vector of indexes returned by the search method
  * @param nnAllDistancesSqr: for each query, squared euclidean distance between the query point and every its neighbor
  * @return true if given indexes and distances are coherent to the groundTruth
  *
  * This method evaluates if the results of the search algorithm are coherent to the groundTruth.
  */
template <typename T>
bool checkCorrectness(std::vector< std::vector<int> > &groundTruth, std::vector<std::vector<int> > &nnAllIndexes, std::vector<std::vector<T> > &nnAllDistancesSqr);

/** Execute the experiments
  */
int main(int argc, char **argv) {
    #ifdef __HIPCC__
        std::cout << "__HIPCC__ defined" << std::endl;
    #endif
    #ifdef _OPENMP
        std::cout << "_OPENMP defined" << std::endl;
    #endif
    // TODO prendere i parametri per gli esperimenti da riga di comando

    std::string dataFolder = "../data";
    std::string experimentsFolder = "../experiments";
    char opt;
    while ((opt = getopt(argc, argv, "d:e:")) != -1) {
        switch (opt) {
            case 'd':
                dataFolder = optarg;
                break;
            case 'e':
                experimentsFolder = optarg;
                break;
            default: /* '?' */
                fprintf(stderr, "Usage: %s [-d path_to_data_folder] [-e path_to_experiments_output_folder]\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    std::cout << "data folder is: " << dataFolder << std::endl;
    std::cout << "output folder is: " << experimentsFolder << std::endl;

    //// files path definition
    // 10^6 examples dataset:
    std::string baseFileName = "/sift/sift_base.fvecs";
    std::string groundtruthFileName = "/sift/sift_groundtruth.ivecs";
    std::string queryFileName = "/sift/sift_query.fvecs";
    // 10^4 examples dataset:
//    std::string baseFileName = "/siftsmall/siftsmall_base.fvecs";
//    std::string groundtruthFileName = "/siftsmall/siftsmall_groundtruth.ivecs";
//    std::string queryFileName = "/siftsmall/siftsmall_query.fvecs";

    // evaluation parameters
    int numResults = 100;

    //// data structures for dataset
    std::vector<std::vector<float> > host_dataset_vv;      // datasetSize x spaceDim       <- dataset where to find nearest neighbors
    std::vector<std::vector<float> > host_queries_vv;      // numQueries  x spaceDim       <- test samples
    std::vector<std::vector<int> > host_grTruth_vv;        // numQueries  x 100            <- first 100 nearest neighbors for each test sample

    //// reading of dataset, queries and groundtruth
    std::cout << "Reading dataset, queries and groundtruth for queries" << std::endl;
    // TODO safely join paths dataFolder and ... (provare con libreria boost)
    bool b = readVecsFile<float, float>(dataFolder + baseFileName, host_dataset_vv, false) &&
        readVecsFile<float, float>(dataFolder + queryFileName, host_queries_vv, false) &&
        readVecsFile<int, int>(dataFolder + groundtruthFileName, host_grTruth_vv, false);

    if(!b){
        std::cerr << "Error: cannot read data" << std::endl;
        return 1;
    }

    // dataset slice (to do quick tests) TODO remove in final version
    const int numExamples = 500;
    //host_dataset_vv = std::vector< std::vector<float> >(host_dataset_vv.begin(), host_dataset_vv.begin() + numExamples);
    host_dataset_vv.resize(numExamples);
    host_dataset_vv.shrink_to_fit();

    //// constants initialization
    const int datasetSize = static_cast<const int>(host_dataset_vv.size());
    std::cout << "Dataset size: " << datasetSize << std::endl;

    const int spaceDim = host_dataset_vv.at(0).size();
    std::cout << "Space size: " << spaceDim << std::endl;

    const int numQueries = static_cast<const int>(host_queries_vv.size());
    std::cout << "Number of queries: " << host_dataset_vv.size() << std::endl;
    std::cout << "k parameter for kNN: " << numResults << std::endl;
    
    b = datasetSize > 0 &&
        spaceDim > 0 &&
        numQueries > 0 &&
        host_queries_vv.size() == host_grTruth_vv.size() &&      // host_queries_vv and host_grTruth_vv must have same length
        numResults <= host_grTruth_vv.at(0).size();            // (numResults <= 100)
    if(!b){
        std::cerr << "Error: invalid sizes/dimensions" << std::endl;
        return 1;
    }

//    // some print to understand data
//    dataPrint(host_dataset_vv, host_grTruth_vv, host_queries_vv);
//    for(int i=0; i<host_grTruth_vv.size(); i++)
//        for(int j=0; j<host_grTruth_vv[0].size(); j++)
//            if(!(host_grTruth_vv[i][j] >= 0 && host_grTruth_vv[i][j] < datasetSize))
//            	cout << "ERRORE: "<< i << " " << j<< " "<< host_grTruth_vv[i][j] << std::endl;


//    // data conversion
//     convert queries from vector of vectors into raw pointer
//     TODO capire perché con la pinned va più lento mentre invece dovrebbe essere più veloce (forse perché il dato trasferito è piccolo, ogni query è 512 byte)
    float *host_queries_ptr;
//    CUDA_CHECK_RETURN(
//            hipHostMalloc((void ** )&host_queries_ptr, sizeof(float) * numQueries * spaceDim)     // allocate pinned memory on host RAM: it allows the use of DMA, speeding up hipMemcpy
//    );
    host_queries_ptr = new float[numQueries * spaceDim]; // non-pinned memory
    for (int i = 0; i < numQueries; i++) {
        // move i-th query from vector of vectors to raw pointer
        std::memcpy(host_queries_ptr + (i * spaceDim), &host_queries_vv[i][0], sizeof(float) * spaceDim);
    }

    //// evaluation: for each implementation, execute search and measure elapsed time
    Search<float> *s;
    auto start = std::chrono::high_resolution_clock::now();
    std::time_t now = std::chrono::system_clock::to_time_t(start);
    std::string strNow = std::ctime(&now);
    std::cout << strNow << std::endl;


    try // I use this for found a exception on csv
    {

        csvfile csv(experimentsFolder + "/ " + strNow + ".csv"); // can throw exception!
        // Header
        csv << "hw" << "num_threads" << "dataset_size" << "init_time" << "eval_time" << "total_time" << endrow;
        // Data example
        // csv <<  "seq" << 0 << 1000 << 0.5 << 0.5 << 1 << endrow;

        //// CPU evaluation
        int maxThreads = 1;
        //TODO use this in final version for compare the dataset length experiment
        //const int datasetLength[] = {10000,50000,150000,450000,1000000};
        const int datasetLength[] = {100,200,300,400,500};
        #ifdef _OPENMP
                maxThreads = omp_get_max_threads()/2;
        #endif
        for(int numCores = 1; numCores <= maxThreads; numCores++){  // openmp directive for the number of cores
            if(numCores < maxThreads && numCores > 1) {
                //TODO add the command for create a csv here like " alg_version;num_threads;dataset_size;time;name "
                std::cout << "Test on CPU, cores: " << numCores << std::endl;
                start = std::chrono::high_resolution_clock::now();
                s = new CpuSearch<float>(host_dataset_vv, numCores);
                std::chrono::duration<double> cpuInitTime = std::chrono::high_resolution_clock::now() - start;
                std::chrono::duration<double> cpuEvalTime = evaluate<float>(s, host_queries_ptr, host_grTruth_vv,
                                                                            numQueries, numResults, true);
                std::cout << "CPU (Cores:" << numCores << ") init time: " << cpuInitTime.count() << std::endl;
                std::cout << "CPU (Cores:" << numCores << ") eval time: " << cpuEvalTime.count() << std::endl;
                csv << "cpu" << numCores << datasetSize << cpuInitTime.count() << cpuEvalTime.count()
                    << cpuInitTime.count() + cpuEvalTime.count() << endrow;
                delete s;

            }
            if(numCores == maxThreads || numCores == 1) {
                for(int n : datasetLength) {
                    std::vector<std::vector<float> > host_dataset_vv_tmp;
                    //host_dataset_vv_tmp = std::vector< std::vector<float> >(host_dataset_vv.begin(), host_dataset_vv.begin() + n);
                    host_dataset_vv_tmp.resize(n);
                    host_dataset_vv_tmp.shrink_to_fit();
                    //MEM COPY
                    std::cout << "Test on CPU, cores: " << numCores << std::endl;
                    start = std::chrono::high_resolution_clock::now();
                    s = new CpuSearch<float>(host_dataset_vv_tmp, numCores);
                    std::chrono::duration<double> cpuInitTime = std::chrono::high_resolution_clock::now() - start;
                    std::chrono::duration<double> cpuEvalTime = evaluate<float>(s, host_queries_ptr, host_grTruth_vv,
                                                                                numQueries, numResults, true);
                    std::cout << "CPU (Cores:" << numCores << ") init time: " << cpuInitTime.count() << std::endl;
                    std::cout << "CPU (Cores:" << numCores << ") eval time: " << cpuEvalTime.count() << std::endl;
                    csv << "cpu" << numCores << n << cpuInitTime.count() << cpuEvalTime.count()
                        << cpuInitTime.count() + cpuEvalTime.count() << endrow;
                    delete s;
                }
            }

        }

        //// GPU evaluation
        #ifdef __HIPCC__
        int cudaBlock [] = {1024,128,4096}; //TODO CHANGE BLOCK IN CONSTRUCTOR FOR CUDA!
        for(int block : cudaBlock){
            for(int n : datasetLength) {
                std::vector<std::vector<float> > host_dataset_vv_tmp;
                host_dataset_vv_tmp.resize(n);
                host_dataset_vv_tmp.shrink_to_fit();

                start = std::chrono::high_resolution_clock::now();
                s = new CudaSearch<float>(host_dataset_vv_tmp,bl);
                std::chrono::duration<double> gpuInitTime = std::chrono::high_resolution_clock::now() - start;
                std::chrono::duration<double> gpuEvalTime = evaluate<float>(s, host_queries_ptr, host_grTruth_vv, numQueries, numResults, true);
                std::cout << "GPU init time: " << gpuInitTime.count() << std::endl;
                std::cout << "GPU eval time: " << gpuEvalTime.count() << std::endl;
                //TODO different block size test
                csv << "gpu" << "BLOCK = xxxx" << n << gpuInitTime.count() << gpuEvalTime.count() << gpuInitTime.count() + gpuEvalTime.count() << endrow;
                delete s;
            }
        }
        #endif
    }
    catch (const std::exception &ex)
    {
        std::cout << "Exception was thrown: " << ex.what() << std::endl;
    }
    // TODO dentro (o dopo) ogni search, salvare risultati su file csv

    delete [] host_queries_ptr;

	return 0;
}

template <typename T>
std::chrono::duration<double> evaluate(Search<T> *s, T* queries_ptr, std::vector <std::vector<int>> &groundTruth, const int &numQueries, int &numResults, bool mustCheckCorrectness){

    //// data structures for query
    std::vector<std::vector<int> > nnAllIndexes(numQueries, std::vector<int>(numResults));    // numQueries x numResults
    std::vector<std::vector<T> > nnAllDistancesSqr(numQueries, std::vector<float>(numResults));     // numQueries x numResults

    // measure time of execution of all queries

    auto start = std::chrono::high_resolution_clock::now();
    int sd = s->getSpaceDim();
    for(int i = 0; i < numQueries; i++){
        s->search(&queries_ptr[i * sd], nnAllIndexes[i], nnAllDistancesSqr[i], numResults);
        //std::cout << vecToStr(nnAllDistancesSqr[i]) << std::endl;
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsedTime = end-start;

    // eventually check for correctness
    if(mustCheckCorrectness) {
        bool c = checkCorrectness(groundTruth, nnAllIndexes, nnAllDistancesSqr);
        std::cout << "correctness: " << (c?"true":"false") << std::endl;
    }

    return elapsedTime;
}

template <typename T>
bool checkCorrectness(std::vector< std::vector<int> > &groundTruth, std::vector<std::vector<int> > &nnAllIndexes, std::vector<std::vector<T> > &nnAllDistancesSqr){
    for(int i = 0; i < nnAllIndexes.size(); i++){
        //std::cout << "\nelement " << i << " of vector groundtruth (" << groundTruth[i].size() << " elements):\n\t" << vecToStr<int>(groundTruth[i]) << std::endl;
        //std::cout << "\nelement " << i << " of vector nnAllIndexes (" << nnAllIndexes[i].size() << " elements):\n\t" << vecToStr<int>(nnAllIndexes[i]) << std::endl;
        if(!checkKNN<float>(groundTruth[i], nnAllIndexes[i], nnAllDistancesSqr[i]))
            return false;
    }
    return true;
}